/*
* Copyright (c) 2023 University of Michigan
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy of this
 * software and associated documentation files (the “Software”), to deal in the Software
 * without restriction, including without limitation the rights to use, copy, modify,
 * merge, publish, distribute, sublicense, and/or sell copies of the Software, and to
 * permit persons to whom the Software is furnished to do so, subject to the following
 * conditions:
 *
 * The above copyright notice and this permission notice shall be included in all copies or
 * substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED “AS IS”, WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED,
 * INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR
 * PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE
 * FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR
 * OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <complex>
#include <hipsolver.h>
#include <green/gpu/cublas_routines_prec.h>

/*hipDoubleComplex ADD(hipDoubleComplex x, hipDoubleComplex y) {
  return hipCadd(x, y);
}
hipComplex ADD(hipComplex x, hipComplex y) {
  return hipCaddf(x, y);
}

hipDoubleComplex SUB(hipDoubleComplex x, hipDoubleComplex y) {
  return hipCsub(x, y);
}
hipComplex SUB(hipComplex x, hipComplex y) {
  return hipCsubf(x, y);
}*/

hipblasStatus_t ASUM(hipblasHandle_t handle, int n,
          const hipDoubleComplex *x, int incx, double *result) {
  return hipblasDzasum(handle, n, x, incx, result);
}
hipblasStatus_t ASUM(hipblasHandle_t handle, int n,
          const hipComplex *x, int incx, float *result) {
  return hipblasScasum(handle, n, x, incx, result);
}


hipblasStatus_t RSCAL(hipblasHandle_t handle, int n, const double *alpha, double *x, int incx) {
  return hipblasDscal(handle, n, alpha, x, incx);
}
hipblasStatus_t RSCAL(hipblasHandle_t handle, int n, const float *alpha, float *x, int incx) {
  return hipblasSscal(handle, n, alpha, x, incx);
}

hipblasStatus_t RAXPY(hipblasHandle_t handle, int n, const double *alpha, const double *x, int incx, double *y, int incy) {
  return hipblasDaxpy(handle, n, alpha, x, incx, y, incy);
}
hipblasStatus_t RAXPY(hipblasHandle_t handle, int n, const float *alpha, const float *x, int incx, float *y, int incy) {
  return hipblasSaxpy(handle, n, alpha, x, incx, y, incy);
}

hipblasStatus_t GEMM(hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb,
        int m, int n, int k,
        const hipDoubleComplex *alpha,
        const hipDoubleComplex *A, int lda,
        const hipDoubleComplex *B, int ldb,
        const hipDoubleComplex *beta,
        hipDoubleComplex *C, int ldc) {
  return hipblasZgemm(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}
hipblasStatus_t GEMM(hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb,
        int m, int n, int k,
        const hipComplex *alpha,
        const hipComplex *A, int lda,
        const hipComplex *B, int ldb,
        const hipComplex *beta,
        hipComplex *C, int ldc) {
  return hipblasCgemm(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}

hipblasStatus_t GEMM_STRIDED_BATCHED(hipblasHandle_t handle,
        hipblasOperation_t transa,
        hipblasOperation_t transb,
        int m, int n, int k,
        const hipDoubleComplex *alpha,
        const hipDoubleComplex *A, int lda,
        long long int          strideA,
        const hipDoubleComplex *B, int ldb,
        long long int          strideB,
        const hipDoubleComplex *beta,
        hipDoubleComplex       *C, int ldc,
        long long int          strideC,
        int batchCount) {
  return hipblasZgemmStridedBatched(handle, transa, transb, m, n, k,
          alpha, A, lda, strideA, B, ldb, strideB, beta, C, ldc, strideC, batchCount);
}
hipblasStatus_t GEMM_STRIDED_BATCHED(hipblasHandle_t handle,
        hipblasOperation_t transa,
        hipblasOperation_t transb,
        int m, int n, int k,
        const hipComplex *alpha,
        const hipComplex *A, int lda,
        long long int          strideA,
        const hipComplex *B, int ldb,
        long long int          strideB,
        const hipComplex *beta,
        hipComplex       *C, int ldc,
        long long int          strideC,
        int batchCount) {
  return hipblasCgemmStridedBatched(handle, transa, transb, m, n, k,
          alpha, A, lda, strideA, B, ldb, strideB, beta, C, ldc, strideC, batchCount);
}

hipblasStatus_t GEAM(hipblasHandle_t handle,
        hipblasOperation_t transa, hipblasOperation_t transb,
        int m, int n,
        const hipDoubleComplex *alpha,
        const hipDoubleComplex *A, int lda,
        const hipDoubleComplex *beta,
        const hipDoubleComplex *B, int ldb,
        hipDoubleComplex *C, int ldc) {
  return hipblasZgeam(handle, transa, transb, m, n, alpha, A, lda, beta, B, ldb, C, ldc);
}
hipblasStatus_t GEAM(hipblasHandle_t handle,
        hipblasOperation_t transa, hipblasOperation_t transb,
        int m, int n,
        const hipComplex *alpha,
        const hipComplex *A, int lda,
        const hipComplex *beta,
        const hipComplex *B, int ldb,
        hipComplex *C, int ldc) {
  return hipblasCgeam(handle, transa, transb, m, n, alpha, A, lda, beta, B, ldb, C, ldc);
}

hipsolverStatus_t POTRF_BATCHED(hipsolverHandle_t handle,
        hipblasFillMode_t uplo,
        int n,
        hipDoubleComplex **Aarray,
        int lda,
        int *infoArray,
        int batchSize) {
  return hipsolverDnZpotrfBatched(handle, uplo, n, Aarray, lda, infoArray, batchSize);
}
hipsolverStatus_t POTRF_BATCHED(hipsolverHandle_t handle,
        hipblasFillMode_t uplo,
        int n,
        hipComplex **Aarray,
        int lda,
        int *infoArray,
        int batchSize) {
  return hipsolverDnCpotrfBatched(handle, uplo, n, Aarray, lda, infoArray, batchSize);
}

hipsolverStatus_t POTRS(hipsolverHandle_t handle,
        hipblasFillMode_t uplo,
        int n,
        int nrhs,
        const hipDoubleComplex *A,
        int lda,
        hipDoubleComplex *B,
        int ldb,
        int *devInfo) {
  return hipsolverDnZpotrs(handle, uplo, n, nrhs, A, lda, B, ldb, devInfo);
}
hipsolverStatus_t POTRS(hipsolverHandle_t handle,
        hipblasFillMode_t uplo,
        int n,
        int nrhs,
        const hipComplex *A,
        int lda,
        hipComplex *B,
        int ldb,
        int *devInfo) {
  return hipsolverDnCpotrs(handle, uplo, n, nrhs, A, lda, B, ldb, devInfo);
}
