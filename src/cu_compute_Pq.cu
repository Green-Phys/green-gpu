#include "hip/hip_runtime.h"
/*
* Copyright (c) 2023 University of Michigan
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy of this
 * software and associated documentation files (the “Software”), to deal in the Software
 * without restriction, including without limitation the rights to use, copy, modify,
 * merge, publish, distribute, sublicense, and/or sell copies of the Software, and to
 * permit persons to whom the Software is furnished to do so, subject to the following
 * conditions:
 *
 * The above copyright notice and this permission notice shall be included in all copies or
 * substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED “AS IS”, WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED,
 * INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR
 * PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE
 * FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR
 * OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include <green/gpu/cugw_qpt.h>

__global__ void validate_info(int *info){
  int idx=blockIdx.x * blockDim.x + threadIdx.x;
  if(idx>0) return;
  if(*info!=0){
    printf("info is: %d\n",*info);
    printf("nonzero info. Aborting application.\n");
    asm("trap;"); // nonzero info = cholesky or LU fails, then all threads should be stopped
  }
}
__global__ void validate_info(int *info, int N){
  int idx=blockIdx.x * blockDim.x + threadIdx.x;
  if(idx>0) return;
  for(int i=0;i<N;++i){
    if(*(info+i)!=0){
      printf("info is: %d\n",*(info+i));
      printf("nonzero info for batched job: %d. Aborting application.\n",i);
      asm("trap;"); // nonzero info = cholesky or LU fails, then all threads should be stopped
    }
  }
}
__global__ void set_up_one_minus_P(hipDoubleComplex *one_minus_P, hipDoubleComplex *P, int naux){
  int  i=blockIdx.x * blockDim.x + threadIdx.x;
  if(i>=naux) return;
  hipDoubleComplex one=make_hipDoubleComplex(1.,0.);
  hipDoubleComplex zero=make_hipDoubleComplex(0.,0.);
  for(int j=0;j<naux;++j){
    one_minus_P[i*naux+j] = hipCsub(zero,P[i*naux+j]);
  }
  one_minus_P[i*naux+i]= hipCadd(one_minus_P[i*naux+i],one);
}
__global__ void set_up_one_minus_P(hipComplex *one_minus_P, hipComplex *P, int naux){
  int  i=blockIdx.x * blockDim.x + threadIdx.x;
  if(i>=naux) return;
  hipComplex one=make_hipComplex(1.,0.);
  hipComplex zero=make_hipComplex(0.,0.);
  for(int j=0;j<naux;++j){
    one_minus_P[i*naux+j] = hipCsubf(zero,P[i*naux+j]);
  }
  one_minus_P[i*naux+i]= hipCaddf(one_minus_P[i*naux+i],one);
}

